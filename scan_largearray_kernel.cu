#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>


#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define TILE_SIZE 1024
// You can use any other block size you wish.
#define BLOCK_SIZE 512


// Host Helper Functions (allocate your own data structure...)


// Device Functions



// Kernel Functions

__global__ void scan(unsigned int *outArray, unsigned int *inArray, unsigned int *sumArray, int numElements){
    
    // shared memory size: 2 * Block_size
    __shared__ unsigned int scanArray[TILE_SIZE];
    int index = blockIdx.x*TILE_SIZE + threadIdx.x;
   
    // Load elements to shared memory
    if(index < numElements && (threadIdx.x!=0 || blockIdx.x!=0))
        scanArray[threadIdx.x] = inArray[index-1];
    else
        scanArray[threadIdx.x] = 0;
    
    if(index+BLOCK_SIZE < numElements)
        scanArray[threadIdx.x + BLOCK_SIZE] = inArray[index-1 + BLOCK_SIZE];
    else
        scanArray[threadIdx.x + BLOCK_SIZE] = 0;


    // prescan operation    
    unsigned int id, stride;
    for(stride=1;stride<TILE_SIZE;stride *= 2){
        __syncthreads();
        id = (threadIdx.x+1) * 2 * stride - 1;
        if(id<TILE_SIZE)
            scanArray[id] += scanArray[id-stride];
    }

    // Post scan
    for( stride=BLOCK_SIZE/2; stride>0; stride /= 2){
        id = (threadIdx.x+1) * 2 * stride - 1;
        if(id + stride < TILE_SIZE)
            scanArray[id+stride] += scanArray[id];
        __syncthreads();
    }
    
    __syncthreads();
    if(threadIdx.x==0)
        sumArray[blockIdx.x] = scanArray[TILE_SIZE-1];
    if(index < numElements)
        outArray[index] = scanArray[threadIdx.x];
    if(index + BLOCK_SIZE < numElements)
        outArray[index+BLOCK_SIZE] = scanArray[threadIdx.x+BLOCK_SIZE]; 
    
}

// Kernel function to perform vector addition of the Auxiliary arrax on the output elements
__global__ void vectorAddition(unsigned int *vector, unsigned int *sumVector, int numElements){
    int index = blockIdx.x*TILE_SIZE + threadIdx.x;
    if(index < numElements){
        vector[index] += sumVector[blockIdx.x];
    }
    if(index + BLOCK_SIZE < numElements){
        vector[index + BLOCK_SIZE] += sumVector[blockIdx.x];
    }
}


// **===-------- Modify the body of this function -----------===**
// You may need to make multiple kernel calls. Make your own kernel
// functions in this file, and then call them from here.
// Note that the code has been modified to ensure numElements is a multiple 
// of TILE_SIZE
void prescanArray(unsigned int *outArray, unsigned int *inArray, int numElements)
{
     
    unsigned int *sumArray;
    int blocks = (int)ceil(numElements/(float)TILE_SIZE);
    hipMalloc((void**) &sumArray, sizeof(unsigned int)*blocks);

    scan<<<blocks, BLOCK_SIZE>>>(outArray, inArray, sumArray, numElements);
    if (blocks > 1) {
        prescanArray(sumArray, sumArray, blocks);

        vectorAddition<<<blocks, BLOCK_SIZE>>>(outArray, sumArray, numElements);
    }
    hipFree(sumArray);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
